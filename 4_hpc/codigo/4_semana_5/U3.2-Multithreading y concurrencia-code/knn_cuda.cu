// knn_cuda.cu
// Brute-force kNN en GPU (L2). Cada hilo procesa 1 query y mantiene un top-k en registros.
// Uso didáctico: para N y Q medianos. Para N muy grande, considerar tiling/BLAS/FAISS.
//
// Compilar: nvcc -O3 -arch=sm_80 -std=c++17 knn_cuda.cu -o knn_cuda

#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <limits>
#include <random>
#include <chrono>
#include <cassert>
#include <cstring>

#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

// ------------ insert_topk<K> y kernel knn_kernel<K> ------------
// Usa INFINITY en lugar de CUDART_INF_F (portátil)
template <int K>
__device__ __forceinline__
void insert_topk(float (&best_dist)[K], int (&best_idx)[K], float d, int idx) {
    if (d >= best_dist[0]) return;
    int pos = 0;
    while (pos < K-1 && best_dist[pos+1] > d) {
        best_dist[pos] = best_dist[pos+1];
        best_idx[pos]  = best_idx[pos+1];
        ++pos;
    }
    best_dist[pos] = d;
    best_idx[pos]  = idx;
}

template <int K>
__global__
void knn_kernel(const float* __restrict__ train,
                const float* __restrict__ query,
                int N, int D, int Q,
                int* __restrict__ out_idx,
                float* __restrict__ out_dist2)
{
    int q = blockIdx.x * blockDim.x + threadIdx.x;
    if (q >= Q) return;

    float best_dist[K];
    int   best_idx[K];
    #pragma unroll
    for (int i = 0; i < K; ++i) { best_dist[i] = INFINITY; best_idx[i] = -1; }

    const float* qptr = query + (size_t)q * D;

    for (int n = 0; n < N; ++n) {
        const float* tptr = train + (size_t)n * D;
        float acc = 0.f;
        int d = 0;
        for (; d + 3 < D; d += 4) {
            float a0 = qptr[d]   - tptr[d];
            float a1 = qptr[d+1] - tptr[d+1];
            float a2 = qptr[d+2] - tptr[d+2];
            float a3 = qptr[d+3] - tptr[d+3];
            acc += a0*a0 + a1*a1 + a2*a2 + a3*a3;
        }
        for (; d < D; ++d) {
            float a = qptr[d] - tptr[d];
            acc += a*a;
        }
        insert_topk<K>(best_dist, best_idx, acc, n);
    }

    int base = q * K;
    for (int i = 0; i < K; ++i) {
        int src = K - 1 - i;
        out_idx[base + i]   = best_idx[src];
        out_dist2[base + i] = best_dist[src];
    }
}

struct Args {
    int N=10000, Q=2000, D=64, K=5, seed=123;
};

Args parse(int argc, char** argv){
    Args a;
    for (int i=1;i<argc;i++){
        if (!strcmp(argv[i],"-N") && i+1<argc) a.N=atoi(argv[++i]);
        else if (!strcmp(argv[i],"-Q") && i+1<argc) a.Q=atoi(argv[++i]);
        else if (!strcmp(argv[i],"-D") && i+1<argc) a.D=atoi(argv[++i]);
        else if (!strcmp(argv[i],"-k") && i+1<argc) a.K=atoi(argv[++i]);
        else if (!strcmp(argv[i],"-seed") && i+1<argc) a.seed=atoi(argv[++i]);
        else if (!strcmp(argv[i],"-h")||!strcmp(argv[i],"--help")){
            printf("Uso: %s -N <train> -Q <query> -D <dim> -k <neighbors> -seed <int>\n", argv[0]);
            exit(0);
        }
    }
    return a;
}

int main(int argc, char** argv){
    auto args = parse(argc, argv);
    int N=args.N, Q=args.Q, D=args.D, K=args.K; // N,Q,D pueden sobreescribirse si luego lees de CSV
    assert(K>=1 && K<=32);

    // Info de dispositivo (para SMs, nombre)
    hipDeviceProp_t prop{}; CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    printf("GPU: %s | SMs=%d | CC=%d.%d\n", prop.name, prop.multiProcessorCount, prop.major, prop.minor);

    printf("KNN mono-GPU | N=%d Q=%d D=%d k=%d\n", N, Q, D, K);

    // Genera datos sintéticos
    std::mt19937 rng(args.seed);
    std::uniform_real_distribution<float> dist(0.0f,1.0f);
    std::vector<float> h_train((size_t)N*D), h_query((size_t)Q*D);
    for (auto &x : h_train) x = dist(rng);
    for (auto &x : h_query) x = dist(rng);

    // Reserva GPU
    float *d_train=nullptr, *d_query=nullptr;
    int   *d_idx=nullptr; float *d_dist2=nullptr;
    CUDA_CHECK(hipMalloc(&d_train,  sizeof(float)* (size_t)N*D));
    CUDA_CHECK(hipMalloc(&d_query,  sizeof(float)* (size_t)Q*D));
    CUDA_CHECK(hipMalloc(&d_idx,    sizeof(int)   * (size_t)Q*K));
    CUDA_CHECK(hipMalloc(&d_dist2,  sizeof(float) * (size_t)Q*K));

    // Copias H2D
    CUDA_CHECK(hipMemcpy(d_train, h_train.data(), sizeof(float)*(size_t)N*D, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_query, h_query.data(), sizeof(float)*(size_t)Q*D, hipMemcpyHostToDevice));

    // Kernel config
    dim3 block(128);
    dim3 grid((Q + block.x - 1) / block.x);

    // Tiempos (solo kernel)
    hipEvent_t evStart, evStop;
    CUDA_CHECK(hipEventCreate(&evStart));
    CUDA_CHECK(hipEventCreate(&evStop));
    CUDA_CHECK(hipEventRecord(evStart));

    switch (K) {
        case 1:  knn_kernel<1><<<grid,block>>>(d_train,d_query,N,D,Q,d_idx,d_dist2); break;
        case 5:  knn_kernel<5><<<grid,block>>>(d_train,d_query,N,D,Q,d_idx,d_dist2); break;
        case 10: knn_kernel<10><<<grid,block>>>(d_train,d_query,N,D,Q,d_idx,d_dist2); break;
        case 16: knn_kernel<16><<<grid,block>>>(d_train,d_query,N,D,Q,d_idx,d_dist2); break;
        case 32: knn_kernel<32><<<grid,block>>>(d_train,d_query,N,D,Q,d_idx,d_dist2); break;
        default: knn_kernel<16><<<grid,block>>>(d_train,d_query,N,D,Q,d_idx,d_dist2); break;
    }
    CUDA_CHECK(hipPeekAtLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(evStop));
    CUDA_CHECK(hipEventSynchronize(evStop));
    float ms_kernel=0.f; CUDA_CHECK(hipEventElapsedTime(&ms_kernel, evStart, evStop));

    // Copia resultados
    std::vector<int>   h_idx((size_t)Q*K);
    std::vector<float> h_dist2((size_t)Q*K);
    CUDA_CHECK(hipMemcpy(h_idx.data(), d_idx, sizeof(int)*(size_t)Q*K, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_dist2.data(), d_dist2, sizeof(float)*(size_t)Q*K, hipMemcpyDeviceToHost));

    // (Mostrar primeras queries si quieres)
    for (int q=0; q < std::min(Q,3); ++q){
        printf("Q[%d]: ", q);
        for (int i=0;i<K;++i){
            printf("(%d, %.4f) ", h_idx[q*K+i], h_dist2[q*K+i]);
        }
        printf("\n");
    }

    // ======================
    // Métricas de performance
    // ======================
    const double t_s = ms_kernel / 1e3;                  // segundos
    const long long pairs = (long long)N * (long long)Q; // comparaciones
    // FLOPs: por dimensión hacemos: 1 resta, 1 multiplicación, 1 suma  => 3 FLOPs
    const double flops = (double)pairs * (double)D * 3.0;
    // Bytes leídos/escritos (modelo ingenuo)
    const double bytes_read  = (double)pairs * (double)D * 2.0 * sizeof(float); // train + query
    const double bytes_write = (double)Q * (double)K * (sizeof(int) + sizeof(float));
    const double bytes_total = bytes_read + bytes_write;

    const double gflops   = flops / 1e9;
    const double gbs      = bytes_total / 1e9;
    const double gflops_s = gflops / t_s;
    const double gbs_s    = gbs    / t_s;

    const long long total_threads = (long long)grid.x * (long long)block.x;
    const double ms_per_query = ms_kernel / (double)Q;
    const double queries_per_s = (double)Q / t_s;

    printf("\n=== Métricas GPU ===\n");
    printf("Grid.x=%d | Block.x=%d | Hilos totales lanzados=%lld\n", grid.x, block.x, total_threads);
    printf("Kernel time: %.3f ms\n", ms_kernel);
    printf("Comparaciones (pairs = N*Q): %lld\n", pairs);
    printf("FLOPs (3*D por par): %.3f GFLOPs | Throughput: %.3f GFLOP/s\n", gflops, gflops_s);
    printf("Bytes (naive): leidos ~ %.3f GB, escritos ~ %.6f GB, total ~ %.3f GB | Throughput: %.3f GB/s\n",
           bytes_read/1e9, bytes_write/1e9, bytes_total/1e9, gbs_s);
    printf("ms/query: %.6f | queries/s: %.2f\n", ms_per_query, queries_per_s);
    printf("SMs GPU: %d\n", prop.multiProcessorCount);

    // Limpieza
    hipFree(d_train); hipFree(d_query);
    hipFree(d_idx);   hipFree(d_dist2);
    hipEventDestroy(evStart); hipEventDestroy(evStop);
    return 0;
}

